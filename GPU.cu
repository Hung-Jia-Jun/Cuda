#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__device__ int add_d(int a, int b)
{
	printf("Hellow world_3\n");
	return a * b;
}

__global__ void add2(int a, int b, int *c)
{
	printf("Hellow world_2\n");
	*c = add_d(a, b);
	printf("Hellow world_4\n");
}
int main2(void)
{
	int c;
	int *ptr;
	hipMalloc((void * *)&ptr, sizeof(int));
	printf("Hellow world_1\n");
	//�iGPU�}�l�p��F
	add2 << < 1, 1 >> > (12, 12, ptr);

	//�Ȯ����CPU�u�{
	hipDeviceSynchronize();

	//�B�槹GPU�A�^��CPU�~���
	printf("Hellow world_5\n");
	hipMemcpy(&c, ptr, sizeof(int), hipMemcpyDeviceToHost);
	printf("Hellow world_6\n");
	printf("12*12 = %d\n", c);
	hipFree(ptr);
	getchar();
	return 0;
}