#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

__global__ void add3(int *a, int *b, int *c)
{
	int tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
	printf("gridDim : %d\n", gridDim.x);
}
int main3(void)
{
	int a[10], b[10], c[10];
	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void * *)&dev_a, 10 * sizeof(int));
	hipMalloc((void * *)&dev_b, 10 * sizeof(int));
	hipMalloc((void * *)&dev_c, 10 * sizeof(int));
	for (int i = 0; i < 10; i++)
	{
		a[i] = i;
		b[i] = i + 2;
	}
	
	//
	hipMemcpy(dev_a/*�ؼЫ��w*/, a/*�����w*/, 10 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b/*�ؼЫ��w*/, b/*�����w*/, 10 * sizeof(int), hipMemcpyHostToDevice);
	add3 << < 1/*�u�{��*/, 10/*�u�{*/ >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c/*�ؼЫ��w*/, dev_c/*�����w*/, 10 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; i++)
	{
		//printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	getchar();
	return 0;

}