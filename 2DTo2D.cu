#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "time.h"
#define N 1024 * 1024

__global__ void add5(int *a, int *b, int *c)
{
	int blockID = blockIdx.x * gridDim.x + blockIdx.x;
	int threadID = blockID * blockDim.x + blockDim.y
		+ threadIdx.y * blockDim.x
		+ threadIdx.x;
	while (threadID < N)
	{
		c[threadID] = a[threadID] + b[threadID];
		threadID += gridDim.x * gridDim.y * blockDim.x * blockDim.y;
	}
}
int main5(void)
{
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	clock_t start, end;
	double duration;
	a = (int *)malloc(N * sizeof(int));
	b = (int *)malloc(N * sizeof(int));
	c = (int *)malloc(N * sizeof(int));

	hipMalloc((void * *)&dev_a, N * sizeof(int));
	hipMalloc((void * *)&dev_b, N * sizeof(int));
	hipMalloc((void * *)&dev_c, N * sizeof(int));

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}

	//dev_a <== a �ƻs��V
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	start = clock();
	dim3 grids(8, 8);
	dim3 threads(8, 8);
	add5 << <grids , threads/*thread*/ >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		if ((a[i] + b[i] == c[i]))
		{
			printf("%d + %d = %d\n", a[i], b[i], c[i]);
		}
	}
	hipDeviceSynchronize();
	end = clock();

	duration = (double)(end - start) / CLOCKS_PER_SEC;
	printf(" %f �� \n", duration);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(a);
	free(b);
	free(c);

	getchar();
	return 0;


}