#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "time.h"
#define N 1024

__global__ void add(int *a, int *b, int *c)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("tid = %d \n", tid);
	c[tid] = a[tid] + b[tid];
	//printf("gridDim.x = %d blockDim.x = %d \n ", gridDim.x , blockDim.x);
	tid += gridDim.x * blockDim.x;
}
int main(void)
{
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	clock_t start, end;
	double duration;
	a = (int *)malloc(N * sizeof(int));
	b = (int *)malloc(N * sizeof(int));
	c = (int *)malloc(N * sizeof(int));

	hipMalloc((void * *)&dev_a, N * sizeof(int));
	hipMalloc((void * *)&dev_b, N * sizeof(int));
	hipMalloc((void * *)&dev_c, N * sizeof(int));

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}

	//dev_a <== a �ƻs��V
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	start = clock();
	add << <10/*block*/, 10/*thread*/ >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		if ((a[i] + b[i] == c[i]))
		{
			printf("%d + %d = %d\n", a[i], b[i], c[i]);
		}
	}
	hipDeviceSynchronize();
	end = clock();

	duration = (double)(end - start) / CLOCKS_PER_SEC;
	//printf(" %f �� \n", duration);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(a);
	free(b);
	free(c);

	getchar();
	return 0;


}